#include "hip/hip_runtime.h"
#include "clutterBoxKernels.cuh"
#include <iostream>

#define GLM_FORCE_CXX98
#include <glm/glm.hpp>
#include <glm/mat4x4.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/matrix_inverse.hpp>

#include <hip/hip_runtime.h>
#include <spinImage/gpu/types/DeviceOrientedPoint.h>
#include <nvidia/hip/hip_runtime_api.h>

// MUST be a define!!!
// Defining it as const float invalidates the results!
#define PI 3.14159265358979323846

__global__ void transformMeshes(glm::mat4* transformations, glm::mat3* normalMatrices, size_t* endIndices, SpinImage::gpu::Mesh scene) {
    size_t threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadIndex >= scene.vertexCount) {
        return;
    }

    unsigned int transformationIndex = 0;
    while(threadIndex >= endIndices[transformationIndex]) {
        transformationIndex++;
    }

    glm::vec4 vertex;
    vertex.x = scene.vertices_x[threadIndex];
    vertex.y = scene.vertices_y[threadIndex];
    vertex.z = scene.vertices_z[threadIndex];
    vertex.w = 1.0;

    glm::vec3 normal;
    normal.x = scene.normals_x[threadIndex];
    normal.y = scene.normals_y[threadIndex];
    normal.z = scene.normals_z[threadIndex];

    glm::vec4 transformedVertex = transformations[transformationIndex] * vertex;
    glm::vec3 transformedNormal = normalMatrices[transformationIndex] * normal;

    transformedNormal = glm::normalize(transformedNormal);

    scene.vertices_x[threadIndex] = transformedVertex.x;
    scene.vertices_y[threadIndex] = transformedVertex.y;
    scene.vertices_z[threadIndex] = transformedVertex.z;

    scene.normals_x[threadIndex] = transformedNormal.x;
    scene.normals_y[threadIndex] = transformedNormal.y;
    scene.normals_z[threadIndex] = transformedNormal.z;

}

void randomlyTransformMeshes(SpinImage::gpu::Mesh scene, std::vector<SpinImage::gpu::Mesh> device_meshList, std::vector<Transformation> transformations) {
    std::vector<size_t> meshEndIndices(device_meshList.size());
    size_t currentEndIndex = 0;

    std::vector<glm::mat4> randomTransformations(device_meshList.size());
    std::vector<glm::mat3> randomNormalTransformations(device_meshList.size());

    for(unsigned int i = 0; i < device_meshList.size(); i++) {
        float yaw = transformations.at(i).rotation.y;
        float pitch = transformations.at(i).rotation.x;
        float roll = transformations.at(i).rotation.z;

        float distanceX = transformations.at(i).position.x;
        float distanceY = transformations.at(i).position.y;
        float distanceZ = transformations.at(i).position.z;

        std::cout << "\t\tRotation: (" << yaw << ", " << pitch << ", "<< roll << "), Translation: (" << distanceX << ", "<< distanceY << ", "<< distanceZ << "), Vertex Count: " << device_meshList.at(i).vertexCount << std::endl;

        glm::mat4 randomRotationTransformation(1.0);
        randomRotationTransformation = glm::rotate(randomRotationTransformation, yaw,   glm::vec3(0, 0, 1));
        randomRotationTransformation = glm::rotate(randomRotationTransformation, pitch, glm::vec3(0, 1, 0));
        randomRotationTransformation = glm::rotate(randomRotationTransformation, roll,  glm::vec3(1, 0, 0));

        glm::mat4 randomTransformation(1.0);
        randomTransformation = glm::translate(randomTransformation, glm::vec3(distanceX, distanceY, distanceZ));
        randomTransformation = randomTransformation * randomRotationTransformation;

        randomTransformations.at(i) = randomTransformation;
        randomNormalTransformations.at(i) = glm::mat3(randomRotationTransformation);

        currentEndIndex += device_meshList.at(i).vertexCount;
        meshEndIndices.at(i) = currentEndIndex;
    }

    glm::mat4* device_transformations;
    size_t transformationBufferSize = device_meshList.size() * sizeof(glm::mat4);
    checkCudaErrors(hipMalloc(&device_transformations, transformationBufferSize));
    checkCudaErrors(hipMemcpy(device_transformations, randomTransformations.data(), transformationBufferSize, hipMemcpyHostToDevice));

    glm::mat3* device_normalMatrices;
    size_t normalMatrixBufferSize = device_meshList.size() * sizeof(glm::mat3);
    checkCudaErrors(hipMalloc(&device_normalMatrices, normalMatrixBufferSize));
    checkCudaErrors(hipMemcpy(device_normalMatrices, randomNormalTransformations.data(), normalMatrixBufferSize, hipMemcpyHostToDevice));

    size_t* device_endIndices;
    size_t startIndexBufferSize = device_meshList.size() * sizeof(size_t);
    checkCudaErrors(hipMalloc(&device_endIndices, startIndexBufferSize));
    checkCudaErrors(hipMemcpy(device_endIndices, meshEndIndices.data(), startIndexBufferSize, hipMemcpyHostToDevice));

    const size_t blockSize = 128;
    size_t blockCount = (scene.vertexCount / blockSize) + 1;
    transformMeshes<<<blockCount, blockSize>>>(device_transformations, device_normalMatrices, device_endIndices, scene);

    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    hipFree(device_transformations);
    hipFree(device_normalMatrices);
    hipFree(device_endIndices);
}

void randomlyTransformMeshes(SpinImage::gpu::Mesh scene, float maxDistance, std::vector<SpinImage::gpu::Mesh> device_meshList, std::minstd_rand0 &randomGenerator) {
    std::uniform_real_distribution<float> distribution(0, 1);

    std::vector<Transformation> transformations;

    for(unsigned int i = 0; i < device_meshList.size(); i++) {
        Transformation trans{};

        trans.rotation.y = float(distribution(randomGenerator) * 2.0 * PI);
        trans.rotation.x = float((distribution(randomGenerator) - 0.5) * PI);
        trans.rotation.z = float(distribution(randomGenerator) * 2.0 * PI);

        trans.position.x = maxDistance * distribution(randomGenerator);
        trans.position.y = maxDistance * distribution(randomGenerator);
        trans.position.z = maxDistance * distribution(randomGenerator);

        transformations.push_back(trans);

    }

    randomlyTransformMeshes(scene, device_meshList, transformations);

}
