#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <cassert>
#include <nvidia/hip/hip_runtime_api.h>
#include "fastPointFeatureHistogramSearcher.cuh"

/*__global__ void calculateSceneAverage(SpinImage::gpu::FPFHHistogram33* histograms, SpinImage::gpu::FPFHHistogram33* averageHistogram, unsigned int count) {
    // Using a running average for better numerical accuracy
    float average = 0;

    for(int i = 0; i < count; i++) {
        average = average + (histograms[i].contents[threadIdx.x] - average) / float(i + 1);
    }

    averageHistogram[threadIdx.x] = average;
}*/

__inline__ __device__ float warpAllReduceSum(float val) {
    for (int mask = warpSize/2; mask > 0; mask /= 2)
        val += __shfl_xor_sync(0xFFFFFFFF, val, mask);
    return val;
}

__inline__ __device__ float computeDescriptorAverage(float* descriptor, unsigned int binsPerHistogram) {
    float threadSum = 0;
    for(unsigned int i = threadIdx.x; i < binsPerHistogram; i += blockDim.x) {
        threadSum += descriptor[i];
    }
    float totalSum = warpAllReduceSum(threadSum);
    return totalSum / float(binsPerHistogram);
}

__device__ float computeFPFHDescriptorSimilarity(
        float* needleDescriptor,
        float needleDescriptorAverage,
        float* haystackDescriptor,
        unsigned int binsPerHistogram) {

    float haystackDescriptorAverage = computeDescriptorAverage(haystackDescriptor, binsPerHistogram);

    float threadSquaredSumX = 0;
    float threadSquaredSumY = 0;
    float threadMultiplicativeSum = 0;

    for(int i = threadIdx.x; i < binsPerHistogram; i += warpSize) {
        float needleDescriptorValue = needleDescriptor[i];
        float haystackDescriptorValue = haystackDescriptor[i];

        float deltaX = float(needleDescriptorValue) - needleDescriptorAverage;
        float deltaY = float(haystackDescriptorValue) - haystackDescriptorAverage;

        threadSquaredSumX += deltaX * deltaX;
        threadSquaredSumY += deltaY * deltaY;
        threadMultiplicativeSum += deltaX * deltaY;
    }

    float squaredSumX = float(sqrt(warpAllReduceSum(threadSquaredSumX)));
    float squaredSumY = float(sqrt(warpAllReduceSum(threadSquaredSumY)));
    float multiplicativeSum = warpAllReduceSum(threadMultiplicativeSum);

    float correlation = multiplicativeSum / (squaredSumX * squaredSumY);

    return correlation;
}


__global__ void computeFPFHSearchResultIndices(
        float* needleDescriptors,
        float* haystackDescriptors,
        unsigned int binsPerHistogram,
        size_t haystackDescriptorCount,
        unsigned int* searchResults) {

#define needleDescriptorIndex blockIdx.x
    assert(blockDim.x == 32);

    extern __shared__ float referenceDescriptor[];

    for(unsigned int i = threadIdx.x; i < binsPerHistogram; i += blockDim.x) {
        referenceDescriptor[i] = needleDescriptors[needleDescriptorIndex * binsPerHistogram + i];
    }

    __syncthreads();

    float referenceDescriptorAverage = computeDescriptorAverage(referenceDescriptor, binsPerHistogram);

    if(referenceDescriptorAverage == 0) {
        if(threadIdx.x == 0) {
            printf("WARNING: descriptor %i consists entirely of zeroes!\n", needleDescriptorIndex);
            // Effectively remove the descriptor from the list of search results
            atomicAdd(&searchResults[needleDescriptorIndex], 1 << 30);
        }
        return;
    }

    float referenceCorrelation = computeFPFHDescriptorSimilarity(
            referenceDescriptor,
            referenceDescriptorAverage,
            haystackDescriptors + binsPerHistogram * needleDescriptorIndex,
            binsPerHistogram);

    // No image pair can have a better correlation than 1, so we can just stop the search right here
    if(referenceCorrelation == 1) {
        return;
    }

    unsigned int searchResultRank = 0;

    for(size_t haystackImageIndex = 0; haystackImageIndex < haystackDescriptorCount; haystackImageIndex++) {
        if(needleDescriptorIndex == haystackImageIndex) {
            continue;
        }

        /*if(blockIdx.x == 0) {
            if(threadIdx.x == 0) {
                printf("%i: ", haystackImageIndex);
            }
            for(unsigned int i = threadIdx.x; i < binsPerHistogram; i += blockDim.x) {
                printf("%f, ", haystackDescriptors[binsPerHistogram * haystackImageIndex + i]);
            }
            if(threadIdx.x == 0) {
                printf("\n");
            }
        }*/

        float correlation = computeFPFHDescriptorSimilarity(
                referenceDescriptor,
                referenceDescriptorAverage,
                haystackDescriptors + binsPerHistogram * haystackImageIndex,
                binsPerHistogram);

        // We've found a result that's better than the reference one. That means this search result would end up
        // above ours in the search result list. We therefore move our search result down by 1.
        if(correlation > referenceCorrelation) {
            searchResultRank++;
        }
    }

    if(threadIdx.x == 0) {
        atomicAdd(&searchResults[needleDescriptorIndex], searchResultRank);
    }
}


SpinImage::array<unsigned int> SpinImage::gpu::computeFPFHSearchResultRanks(
        SpinImage::gpu::FPFHHistograms device_needleDescriptors,
        size_t needleDescriptorCount,
        SpinImage::gpu::FPFHHistograms device_haystackDescriptors,
        size_t haystackDescriptorCount,
        SpinImage::debug::FPFHSearchRunInfo* runInfo) {

    auto executionStart = std::chrono::steady_clock::now();

    size_t searchResultBufferSize = needleDescriptorCount * sizeof(unsigned int);
    unsigned int* device_searchResults;
    checkCudaErrors(hipMalloc(&device_searchResults, searchResultBufferSize));
    checkCudaErrors(hipMemset(device_searchResults, 0, searchResultBufferSize));

    const unsigned int binsPerHistogram = 3 * device_needleDescriptors.binsPerHistogramFeature;
    size_t singleHistogramSizeBytes = binsPerHistogram * sizeof(float);

    auto searchStart = std::chrono::steady_clock::now();


    computeFPFHSearchResultIndices<<<needleDescriptorCount, 32, singleHistogramSizeBytes>>>(
         device_needleDescriptors.histograms,
         device_haystackDescriptors.histograms,
         binsPerHistogram,
         haystackDescriptorCount,
         device_searchResults);

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    std::chrono::milliseconds searchDuration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - searchStart);

    array<unsigned int> resultIndices;
    resultIndices.content = new unsigned int[needleDescriptorCount];
    resultIndices.length = needleDescriptorCount;

    checkCudaErrors(hipMemcpy(resultIndices.content, device_searchResults, searchResultBufferSize, hipMemcpyDeviceToHost));

    // Cleanup
    checkCudaErrors(hipFree(device_searchResults));

    std::chrono::milliseconds executionDuration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - executionStart);

    if(runInfo != nullptr) {
        runInfo->searchExecutionTimeSeconds = double(searchDuration.count()) / 1000.0;
        runInfo->totalExecutionTimeSeconds = double(executionDuration.count()) / 1000.0;
    }

    return resultIndices;
}



