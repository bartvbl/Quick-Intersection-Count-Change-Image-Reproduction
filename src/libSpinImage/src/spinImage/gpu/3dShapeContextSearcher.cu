#include "hip/hip_runtime.h"
#include <algorithm>
#include <chrono>
#include "3dShapeContextSearcher.cuh"
#include <nvidia/hip/hip_runtime_api.h>
#include <cfloat>
#include <host_defines.h>
#include <iostream>
#include <hip/hip_vector_types.h>

const size_t elementsPerShapeContextDescriptor =
        SHAPE_CONTEXT_HORIZONTAL_SLICE_COUNT *
        SHAPE_CONTEXT_VERTICAL_SLICE_COUNT *
        SHAPE_CONTEXT_LAYER_COUNT;


__inline__ __device__ float warpAllReduceSum(float val) {
    for (int mask = warpSize/2; mask > 0; mask /= 2)
        val += __shfl_xor_sync(0xFFFFFFFF, val, mask);
    return val;
}

__inline__ __device__ float warpAllReduceMin(float val) {
    for (int mask = warpSize/2; mask > 0; mask /= 2)
        val = min(__shfl_xor_sync(0xFFFFFFFF, val, mask), val);
    return val;
}

__device__ float compute3DSCPairDistanceGPU(
        shapeContextBinType* needleDescriptor,
        shapeContextBinType* haystackDescriptor,
        float* sharedSquaredSums) {

#define sliceOffset threadIdx.y
    float threadSquaredDistance = 0;
    for(short binIndex = threadIdx.x; binIndex < elementsPerShapeContextDescriptor; binIndex += blockDim.x) {
        float needleBinValue = needleDescriptor[binIndex];
        short haystackBinIndex =
            (binIndex + (sliceOffset * SHAPE_CONTEXT_VERTICAL_SLICE_COUNT * SHAPE_CONTEXT_LAYER_COUNT));
        // Simple modulo that I think is less expensive
        if(haystackBinIndex >= elementsPerShapeContextDescriptor) {
            haystackBinIndex -= elementsPerShapeContextDescriptor;
        }
        float haystackBinValue = haystackDescriptor[haystackBinIndex];
        float binDelta = needleBinValue - haystackBinValue;
        threadSquaredDistance += binDelta * binDelta;
    }

    float combinedSquaredDistance = warpAllReduceSum(threadSquaredDistance);

    if(threadIdx.x == 0) {
        sharedSquaredSums[sliceOffset] = combinedSquaredDistance;
    }

    __syncthreads();

    // An entire warp must participate in the reduction, so we give the excess threads
    // the highest possible value so that any other value will be lower
    float threadValue = threadIdx.x < SHAPE_CONTEXT_HORIZONTAL_SLICE_COUNT ?
            sharedSquaredSums[threadIdx.x] : FLT_MAX;
    float lowestDistance = std::sqrt(warpAllReduceMin(threadValue));

    // Some threads will race ahead to the next image pair. Need to avoid that.
    __syncthreads();

    return lowestDistance;
}

__global__ void computeShapeContextSearchResultIndices(
        shapeContextBinType* needleDescriptors,
        shapeContextBinType* haystackDescriptors,
        size_t haystackDescriptorCount,
        float haystackScaleFactor,
        unsigned int* searchResults) {
#define needleDescriptorIndex blockIdx.x

    // Since memory is reused a lot, we cache both the needle and haystack image in shared memory
    // Combined this is is approximately (at default settings) the size of a spin or RICI image

    __shared__ shapeContextBinType referenceDescriptor[elementsPerShapeContextDescriptor];
    for(unsigned int index = blockDim.x * threadIdx.y + threadIdx.x; index < elementsPerShapeContextDescriptor; index += blockDim.x * blockDim.y) {
        referenceDescriptor[index] = needleDescriptors[elementsPerShapeContextDescriptor * needleDescriptorIndex + index];
    }

    __shared__ shapeContextBinType haystackDescriptor[elementsPerShapeContextDescriptor];
    for(unsigned int index = blockDim.x * threadIdx.y + threadIdx.x; index < elementsPerShapeContextDescriptor; index += blockDim.x * blockDim.y) {
        haystackDescriptor[index] =
                haystackDescriptors[elementsPerShapeContextDescriptor * needleDescriptorIndex + index]
                * (1.0f/haystackScaleFactor);
    }

    __shared__ float squaredSums[SHAPE_CONTEXT_HORIZONTAL_SLICE_COUNT];

    __syncthreads();

    float referenceDistance = compute3DSCPairDistanceGPU(
            referenceDescriptor,
            haystackDescriptor,
            squaredSums);

    // No image pair can have a better distance than 0, so we can just stop the search right here
    if(referenceDistance == 0) {
        return;
    }

    unsigned int searchResultRank = 0;

    for(size_t haystackDescriptorIndex = 0; haystackDescriptorIndex < haystackDescriptorCount; haystackDescriptorIndex++) {
        if(needleDescriptorIndex == haystackDescriptorIndex) {
            continue;
        }

        for(unsigned int index = blockDim.x * threadIdx.y + threadIdx.x; index < elementsPerShapeContextDescriptor; index += blockDim.x * blockDim.y) {
            haystackDescriptor[index] =
                    haystackDescriptors[elementsPerShapeContextDescriptor * haystackDescriptorIndex + index]
                    * (1.0f/haystackScaleFactor);
        }

        __syncthreads();

        float distance = compute3DSCPairDistanceGPU(
                referenceDescriptor,
                haystackDescriptor,
                squaredSums);

        // We've found a result that's better than the reference one. That means this search result would end up
        // above ours in the search result list. We therefore move our search result down by 1.
        if(distance < referenceDistance) {
            searchResultRank++;
        }
    }

    if(threadIdx.x == 0) {
        searchResults[needleDescriptorIndex] = searchResultRank;
    }
}



SpinImage::array<unsigned int> SpinImage::gpu::compute3DSCSearchResultRanks(
        array<shapeContextBinType> device_needleDescriptors,
        size_t needleDescriptorCount,
        size_t needleDescriptorSampleCount,
        array<shapeContextBinType> device_haystackDescriptors,
        size_t haystackDescriptorCount,
        size_t haystackDescriptorSampleCount,
        SpinImage::debug::SCSearchRunInfo* runInfo) {
    static_assert(SHAPE_CONTEXT_HORIZONTAL_SLICE_COUNT <= 32);

    auto executionStart = std::chrono::steady_clock::now();

    size_t searchResultBufferSize = needleDescriptorCount * sizeof(unsigned int);
    unsigned int* device_searchResults;
    checkCudaErrors(hipMalloc(&device_searchResults, searchResultBufferSize));
    checkCudaErrors(hipMemset(device_searchResults, 0, searchResultBufferSize));

    float haystackScaleFactor = float(double(needleDescriptorSampleCount) / double(haystackDescriptorSampleCount));
    std::cout << "\t\tHaystack scale factor: " << haystackScaleFactor << std::endl;

    auto searchStart = std::chrono::steady_clock::now();

    dim3 blockDimensions = {
        32, SHAPE_CONTEXT_HORIZONTAL_SLICE_COUNT, 1
    };
    computeShapeContextSearchResultIndices<<<needleDescriptorCount, blockDimensions>>>(
        device_needleDescriptors.content,
        device_haystackDescriptors.content,
        haystackDescriptorCount,
        haystackScaleFactor,
        device_searchResults);

    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipGetLastError());

    std::chrono::milliseconds searchDuration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - searchStart);

    array<unsigned int> resultIndices;
    resultIndices.content = new unsigned int[needleDescriptorCount];
    resultIndices.length = needleDescriptorCount;

    checkCudaErrors(hipMemcpy(resultIndices.content, device_searchResults, searchResultBufferSize, hipMemcpyDeviceToHost));

    // Cleanup

    hipFree(device_searchResults);

    std::chrono::milliseconds executionDuration = std::chrono::duration_cast<std::chrono::milliseconds>(std::chrono::steady_clock::now() - executionStart);

    if(runInfo != nullptr) {
        runInfo->searchExecutionTimeSeconds = double(searchDuration.count()) / 1000.0;
        runInfo->totalExecutionTimeSeconds = double(executionDuration.count()) / 1000.0;
    }

    return resultIndices;
}